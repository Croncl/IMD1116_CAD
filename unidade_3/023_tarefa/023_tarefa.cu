#include <stdio.h>
#include <hip/hip_runtime.h>

#define NX 300
#define NY 300
#define NZ 300
#define NSTEPS 2000

const double nu = 0.1;
const double dt = 0.001;

__global__ void atualiza(double *u, double *u_new, int nx, int ny, int nz, double nu, double dt) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // índice linear na ordem i*NY*NZ + j*NZ + k
    if (x >= 1 && x < nx-1 && y >= 1 && y < ny-1 && z >= 1 && z < nz-1) {
        int idx = x * ny * nz + y * nz + z;

        // índices dos vizinhos
        int idx_xm = (x-1) * ny * nz + y * nz + z;
        int idx_xp = (x+1) * ny * nz + y * nz + z;
        int idx_ym = x * ny * nz + (y-1) * nz + z;
        int idx_yp = x * ny * nz + (y+1) * nz + z;
        int idx_zm = x * ny * nz + y * nz + (z-1);
        int idx_zp = x * ny * nz + y * nz + (z+1);

        // cálculo da atualização pelo esquema explícito
        double laplaciano = (u[idx_xp] + u[idx_xm] + u[idx_yp] + u[idx_ym] + u[idx_zp] + u[idx_zm] - 6.0 * u[idx]) / (1.0 * 1.0);
        u_new[idx] = u[idx] + nu * dt * laplaciano;
    }
}

int main() {
    size_t size = NX * NY * NZ * sizeof(double);
    double *h_u = (double*)malloc(size);
    double *d_u, *d_u_new;

    // Inicializa condição inicial: 0.0 em todo lugar, exceto centro com 100.0
    for (int i=0; i<NX; i++) {
        for (int j=0; j<NY; j++) {
            for (int k=0; k<NZ; k++) {
                int idx = i * NY * NZ + j * NZ + k;
                h_u[idx] = 0.0;
            }
        }
    }
    int cx = NX / 2;
    int cy = NY / 2;
    int cz = NZ / 2;
    h_u[cx * NY * NZ + cy * NZ + cz] = 100.0;

    hipMalloc(&d_u, size);
    hipMalloc(&d_u_new, size);
    hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);

    dim3 threads(8,8,8);
    dim3 blocks((NX + threads.x - 1)/threads.x,
                (NY + threads.y - 1)/threads.y,
                (NZ + threads.z - 1)/threads.z);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int step = 0; step < NSTEPS; step++) {
        atualiza<<<blocks, threads>>>(d_u, d_u_new, NX, NY, NZ, nu, dt);
        hipDeviceSynchronize();

        // troca ponteiros
        double *temp = d_u;
        d_u = d_u_new;
        d_u_new = temp;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // copia resultado final para host
    hipMemcpy(h_u, d_u, size, hipMemcpyDeviceToHost);

    // imprime valor central
    printf("Valor no centro após %d passos: %f\n", NSTEPS, h_u[cx * NY * NZ + cy * NZ + cz]);
    printf("Tempo total CUDA: %f ms\n", milliseconds);

    // libera memória
    hipFree(d_u);
    hipFree(d_u_new);
    free(h_u);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}


/*
#!/bin/bash
#SBATCH --partition=gpu-4-a100        # Partição com GPUs V100/A100
#SBATCH --gpus-per-node=1             # 1 GPU por nó
#SBATCH --nodes=1                     # 1 nó
#SBATCH --time=00:10:00               # Tempo máximo (10 minutos)
#SBATCH --job-name=023_tarefa_cuda    # Nome do job
#SBATCH --output=023_tarefa_cuda_%j.out
#SBATCH --error=023_tarefa_cuda_%j.err

module load compilers/nvidia/cuda/12.6
module load compilers/gnu/11.2

# Compilar o código CUDA
nvcc 023_tarefa_cuda.cu -o 023_tarefa_cuda -lm

echo "Iniciando simulação CUDA 3D com perfilamento NSYS..."

# Executar a simulação com perfilamento NSYS, saída ficará no .out padrão
nsys profile --stats=true -o 023_tarefa_cuda_profile_${SLURM_JOB_ID} ./023_tarefa_cuda

echo "========================================"
echo "Simulação e perfilamento concluídos."
echo "- Perfil NSYS salvo em: 023_tarefa_cuda_profile_${SLURM_JOB_ID}.qdrep"
echo "Use Nsight Systems GUI para abrir e analisar o perfil."


*/

