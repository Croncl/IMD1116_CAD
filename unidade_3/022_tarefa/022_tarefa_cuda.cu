#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NX 300
#define NY 300
#define NZ 300
#define NSTEPS 2000

const double nu = 0.1;
const double dt = 0.01;
const double dx = 1.0;
const double alpha = nu * dt / (dx * dx);

__global__ void atualiza(double* vnew, double* vold, int nx, int ny, int nz, double alpha) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (x > 0 && x < nx-1 && y > 0 && y < ny-1 && z > 0 && z < nz-1) {
        int idx = z * ny * nx + y * nx + x;
        double laplacian = (vold[idx+1] + vold[idx-1] +
                          vold[idx+nx] + vold[idx-nx] +
                          vold[idx+nx*ny] + vold[idx-nx*ny] - 
                          6.0 * vold[idx]) / (dx * dx);
        vnew[idx] = vold[idx] + dt * nu * laplacian;
    }
}

void simulate_cuda(double *h_u, const char *filename) {
    double *d_u, *d_u_new;
    size_t size = NX * NY * NZ * sizeof(double);
    
    hipMalloc(&d_u, size);
    hipMalloc(&d_u_new, size);
    hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);

    FILE *fp = fopen(filename, "w");
    fprintf(fp, "Passo,ValorCentral,Tempo\n");

    int bx = 8, by = 8, bz = 8;
    dim3 threads(bx, by, bz);
    dim3 grid((NX+bx-1)/bx, (NY+by-1)/by, (NZ+bz-1)/bz);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int n = 0; n < NSTEPS; ++n) {
        hipEventRecord(start);
        atualiza<<<grid, threads>>>(d_u_new, d_u, NX, NY, NZ, alpha);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms;
        hipEventElapsedTime(&ms, start, stop);
        double elapsed_step = ms / 1000.0;

        double *tmp = d_u;
        d_u = d_u_new;
        d_u_new = tmp;

        // Get center value
        double center_value;
        int center_idx = (NX/2) * NY * NZ + (NY/2) * NZ + (NZ/2);
        hipMemcpy(&center_value, &d_u[center_idx], sizeof(double), hipMemcpyDeviceToHost);
        
        fprintf(fp, "%d,%.5f,%.9f\n", n, center_value, elapsed_step);

        if (n % 200 == 0) {
            printf("[CUDA] Passo %d | Valor central: %.5f | Tempo passo: %.6f s\n",
                  n, center_value, elapsed_step);
        }
    }

    hipMemcpy(h_u, d_u, size, hipMemcpyDeviceToHost);
    fclose(fp);
    hipFree(d_u);
    hipFree(d_u_new);
}

int main() {
    printf("Iniciando simulação CUDA 3D\n");
    
    int size = NX * NY * NZ;
    double *h_u = (double*)malloc(size * sizeof(double));
    
    // Initialize with central peak
    for (int i = 0; i < NX; ++i)
        for (int j = 0; j < NY; ++j)
            for (int k = 0; k < NZ; ++k) {
                if ((i > NX/2-2 && i < NX/2+2) &&
                    (j > NY/2-2 && j < NY/2+2) &&
                    (k > NZ/2-2 && k < NZ/2+2))
                    h_u[i*NY*NZ + j*NZ + k] = 1.0;
                else
                    h_u[i*NY*NZ + j*NZ + k] = 0.0;
            }

    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    simulate_cuda(h_u, "simulacao_cuda.csv");
    
    clock_gettime(CLOCK_MONOTONIC, &end);
    double elapsed = (end.tv_sec - start.tv_sec) + 
                   (end.tv_nsec - start.tv_nsec) / 1e9;
    
    printf("Tempo total CUDA: %.3f segundos\n", elapsed);
    free(h_u);
    return 0;
}

/*
#!/bin/bash
#SBATCH --partition=gpu-8-v100
#SBATCH --gpus-per-node=1
#SBATCH --nodes=1
#SBATCH --time=00:10:00
#SBATCH --job-name=022_tarefa_cuda
#SBATCH --output=022_tarefa_cuda_%j.out
#SBATCH --error=022_tarefa_cuda_%j.err

module load compilers/nvidia/cuda/12.6
module load compilers/gnu/14.2.0

# Compilar
nvcc 022_tarefa_cuda.cu -o 022_tarefa_cuda -lm

# Executar e mover resultados
./022_tarefa_cuda
mv simulacao_cuda.csv "simulacao_cuda_${SLURM_JOB_ID}.csv"

echo "Simulação concluída. Resultados em simulacao_cuda_${SLURM_JOB_ID}.csv"

*/